#include "hip/hip_runtime.h"
/**
 * 
 * @authors
 * Jaswanth Yenduri (jyendur@siue.edu)  800746158
 * Likitha Vinjam   (lvinjam@siue.edu)  800748958
 * Manisha Reddy Tummala  (mtummal@siue.edu) 800722182
 * 
 */

/*
 * @file multithread.cu
*/

#include <iostream>
#include <sstream>
#include <fstream>
#include <string>
using namespace std;

/**********************************************************
***********************************************************
*               error checking stufff
***********************************************************
***********************************************************/
// Enable this for error checking

#define CUDA_CHECK_ERROR
#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError() __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line ) {

    #ifdef CUDA_CHECK_ERROR

    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    do {
        if ( hipSuccess != err ) {
            fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n", file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );
    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}

inline void __cudaCheckError( const char *file, const int line ) {

    #ifdef CUDA_CHECK_ERROR
    #pragma warning( push )
    #pragma warning( disable: 4127 ) // Prevent warning on do-while(0);
    do {
        hipError_t err = hipGetLastError();
        if ( hipSuccess != err ) {
            fprintf( stderr, "cudaCheckError() failed at %s:%i : %s.\n", file, line, hipGetErrorString( err ) );
            exit( -1 );
        }

        // More careful checking. However, this will affect performance.
        // Comment if not needed.
        err = hipDeviceSynchronize();
        if( hipSuccess != err ) {
            fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s.\n", file, line, hipGetErrorString( err ) );
            exit( -1 );
        }
    } while ( 0 );

    #pragma warning( pop )
    #endif // CUDA_CHECK_ERROR
    return;
}
/***************************************************************
* **************************************************************
* end of error checking stuff
****************************************************************
***************************************************************/

const int blockSize = 1024;

// function takes an array pointer, and the number of rows and cols in the array, and
// allocates and intializes the array to a bunch of random numbers
// Note that this function creates a 1D array that is a flattened 2D array
// to access data item data[i][j], you must can use data[(i*rows) + j]
int * makeRandArray( const int size, const int seed ) {
    srand( seed );
    int * array = new int[ size ];

    for( int i = 0; i < size; i ++ )
        array[i] = std::rand() % 1000000;

    return array;
}

//*******************************//
//your kernel here!!!!!!!!!!!!!!!!!
//*******************************//
__global__ void kernel_findMax(const int* device_array, int size, int* output) {
    
    int th_index = threadIdx.x;
    int index = threadIdx.x + blockIdx.x * blockSize;
    const int gridSize = blockSize * gridDim.x;
    int sum = 0;

    for (int i = index; i < size; i += gridSize)
        sum += device_array[i];

    __shared__ int cache[blockSize];
    cache[th_index] = sum;
    __syncthreads();

    int temp = blockDim.x / 2;
    while(temp > 0) {
        if(th_index < temp && cache[th_index] < cache[th_index + temp])
            cache[th_index] = cache[th_index + temp];
        __syncthreads();

        temp = temp/2;
    }

    if (th_index == 0) 
        output[blockIdx.x] = cache[0];
}

__global__ void kernel_countSort(int* device_array,int *device_count,int size,int exp) {

    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    if (idx < size)
        atomicAdd(&(device_count[(device_array[idx] / exp) % 10]), 1);
    else
        return;
}

__global__ void kernel_outputToArray(int* device_array, int* device_output, int size) {

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < size)
        device_array[tid] = device_output[tid];
    else
        return;
}

void radixSort(int *array,int *output,int *count,int size, int digit) {

    for (int i = 1; i < 10; i++)
        count[i] += count[i - 1];

    for (int i = size - 1; i >= 0; i--)
    {
        output[count[(array[i] / digit) % 10] - 1] = array[i];
        count[(array[i] / digit) % 10]--;
    }
}

int main(int argc, char* argv[]) {

    int * array; // the poitner to the array of rands
    int size, seed; // values for the size of the array and the seed for generating random numbers
    bool printSorted = false;

    // check the command line args
    if( argc < 4 ){
        std::cerr << "usage: " << argv[0]
                    << " [amount of random nums to generate] [seed value for rand]"
                    << " [1 to print sorted array, 0 otherwise]" 
                    << std::endl;
        exit( -1 );
    }

    // convert cstrings to ints
    {
        std::stringstream ss1( argv[1] );
        ss1 >> size;
    }

    {
        std::stringstream ss1( argv[2] );
        ss1 >> seed;
    }

    {
        int sortPrint;
        std::stringstream ss1( argv[3] );
        ss1 >> sortPrint;
        if( sortPrint == 1 )
        printSorted = true;
    }

    // get the random numbers
    array = makeRandArray( size, seed );

    int * device_array;
    int * device_count;
    int * device_output;
    int * device_max;

    int *count;
    int *output;
    int* max;

    output = (int*)malloc(size * sizeof(int));
    count = (int*)malloc(10 * sizeof(int));
    max = (int*)malloc(sizeof(int));    

    if( printSorted ){
        ///////////////////////////////////////////////
        /// Your code to print the sorted array here //
        ///////////////////////////////////////////////

        for( int i = 0; i < size; i ++ )
            cout << array[i] << " ";
        cout << endl;
    }

    /***********************************
    * create a cuda timer to time execution
    **********************************/
    hipEvent_t startTotal, stopTotal;
    float timeTotal;
    hipEventCreate(&startTotal);
    hipEventCreate(&stopTotal);
    hipEventRecord( startTotal, 0 );
    /***********************************
    * end of cuda timer creation
    **********************************/

    /////////////////////////////////////////////////////////////////////
    ///////////////////////    YOUR CODE HERE     ///////////////////////
    /////////////////////////////////////////////////////////////////////
    /*
    * You need to implement your kernel as a function at the top of this file.
    * Here you must
    * 1) allocate device memory
    * 2) set up the grid and block sizes
    * 3) call your kenrnel
    * 4) get the result back from the GPU
    *
    *
    * to use the error checking code, wrap any cudamalloc functions as follows:
    * CudaSafeCall( hipMalloc( &pointer_to_a_device_pointer,
    * length_of_array * sizeof( int ) ) );
    * Also, place the following function call immediately after you call your kernel
    * ( or after any other cuda call that you think might be causing an error )
    * CudaCheckError();
    */

    CudaSafeCall ( hipMalloc( &device_array,size * sizeof(int)) );
    hipMemcpy(device_array, array, size * sizeof(int), hipMemcpyHostToDevice);
    CudaCheckError();

    CudaSafeCall ( hipMalloc( &device_output,size * sizeof(int)) );
    CudaSafeCall ( hipMalloc( &device_count, 10 * sizeof(int)) );
    CudaSafeCall ( hipMalloc( &device_max, sizeof(int)) );

    //dim3 threadsPerBlock(size + 1023 / blockSize);
    //dim3 numBlocks(blockSize);

    dim3 threadsPerBlock( 1024 );
    dim3 numBlocks( ceil((size)/(float)1024) + 1 );

    //cout << blockSize << " " << threadsPerBlock << endl;
    
    kernel_findMax << <numBlocks, threadsPerBlock>> > (device_array, size, device_max);
    hipMemcpy(max, device_max, sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 1; *max / i > 0; i *= 10)
    {
        hipMemset(device_count, 0, 10 * sizeof(int));
        kernel_countSort <<< numBlocks, threadsPerBlock >>> (device_array, device_count, size, i);

        hipMemcpy(count, device_count, 10 * sizeof(int), hipMemcpyDeviceToHost);
        CudaCheckError();

        radixSort(array, output, count ,size, i);

        hipMemcpy(device_output, output, size * sizeof(int), hipMemcpyHostToDevice);
        CudaCheckError();
    
        //kernel_outputToArray << <numBlocks, threadsPerBlock >> > (device_array, device_output, size);

        hipMemcpy(array, device_output, size * sizeof(int), hipMemcpyDeviceToHost);
        CudaCheckError();
    }

    /***********************************
    * Stop and destroy the cuda timer
    **********************************/
    hipEventRecord( stopTotal, 0 );
    hipEventSynchronize( stopTotal );
    hipEventElapsedTime( &timeTotal, startTotal, stopTotal );
    hipEventDestroy( startTotal );
    hipEventDestroy( stopTotal );
    /***********************************
    * end of cuda timer destruction
    **********************************/

   hipFree(device_array);

    std::cerr << "Total time in seconds: "
    << timeTotal / 1000.0 << std::endl;


    if( printSorted ){
        ///////////////////////////////////////////////
        /// Your code to print the sorted array here //
        ///////////////////////////////////////////////

        for( int i = 0; i < size; i ++ )
            cout << array[i] << " ";
        cout << endl;
    }

    hipFree(device_array);
    hipFree(device_output);
    hipFree(device_count);
    hipFree(device_max);

    free(array);
    free(output);
    free(count);
    free(max);
    return 0;
}